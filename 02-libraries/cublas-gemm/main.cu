/* Includes, system */
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <iostream>
#include <iomanip>
#include <random>
#include <cblas.h>
#include <omp.h>

#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <hip/hip_fp16.h>

#define LIM_CHECK_N 4096
#define LIM_PRINT_N 32
// fraction error   1.0 is 100% 
#define TOLERR 0.0001
#ifdef CPUFP64
    typedef double CPUTYPE;
#else
    typedef float CPUTYPE;
#endif
#include "tools.h"

using namespace std;

int main(int argc, char **argv) {
  hipblasStatus_t status;
  if(argc != 6){
      fprintf(stderr, "run as ./prog dev nt n comptype mode\n\n"
              "dev:      Device ID\n"
              "nt:       Number of CPU threads (accelerates data init and CPU mode)\n"
              "n:        Matrix size of n x n\n"
              "comptype: GPU CUBLAS mode\n"
              "mode:     CPU=0,  GPU=1\n\n");

      printArgsInfo();
      return EXIT_FAILURE;
  }
  float gputime_ms;
  int dev = atoi(argv[1]);
  int nt = atoi(argv[2]);
  int N = atoi(argv[3]);
  int comptype = atoi(argv[4]);
  int mode = atoi(argv[5]);
  printf("\n*********************************************\n"
         "******** CUBLAS Example by Temporal *********\n"
         "*********************************************\n\n");
  printf("dev=%i, nt=%i, n=%i, cublasType=%i, <mode = %i -> %s>\n\n", dev, nt, N, comptype, mode, mode == 0? "CPU" : "GPU");
  // host pointers
  ATYPE *h_A;
  BTYPE *h_B;
  CTYPE *h_C;
  CPUTYPE *cblasC;
  // device pointers
  ATYPE *d_A = 0;
  BTYPE *d_B = 0;
  CTYPE *d_C = 0;
  // constants
  CTYPE alpha = 1.0f;
  CTYPE beta = 0.0f;
  // number of elements
  unsigned long nelem = (unsigned long)N * (unsigned long)N;
  double GBytesUsed = (double)nelem*(sizeof(ATYPE) + sizeof(BTYPE) + sizeof(CTYPE))/1e9;
  double t1, t2;
  double TFLOP = 2.0*(double)N*(double)N*(double)N * 1E-12;
  int bitsA = sizeof(ATYPE)*8;
  int bitsB = sizeof(BTYPE)*8;
  int bitsC = sizeof(CTYPE)*8;
  int bitsCPU = sizeof(CPUTYPE)*8;

  hipDataType dtypeA = dataTypes[hmap(bitsA)];
  hipDataType dtypeB = dataTypes[hmap(bitsB)];
  hipDataType dtypeC = dataTypes[hmap(bitsC)];
  const char* dtypeAStr = dataTypesStr[hmap(bitsA)];
  const char* dtypeBStr = dataTypesStr[hmap(bitsB)];
  const char* dtypeCStr = dataTypesStr[hmap(bitsC)];
  const char* dtypeCPU = cblasDataTypesStr[cpuhmap(bitsCPU)];

  gpuErrchk(hipSetDevice(dev));
  print_gpu_specs(dev);
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  hipblasHandle_t handle;
  omp_set_num_threads(nt);
  printf("Matrix size %i x %i --> %lu elements\n"
          "GPU: A FP%i (%10s), B FP%i (%10s), C FP%i (%10s)\n"
          "CPU: A FP%i (%10s), B FP%i (%10s), C FP%i (%10s)\n\n", 
          N, N, nelem,  
          bitsA, dtypeAStr,
          bitsB, dtypeBStr,
          bitsC, dtypeCStr, 
          bitsCPU, dtypeCPU,
          bitsCPU, dtypeCPU,
          bitsCPU, dtypeCPU);

  printf("GPU Mem used...................%f GB\n", GBytesUsed); fflush(stdout);
  printf("Pinned Mem.....................");
  #ifdef PINNED
    printf("True\n");
  #else
    printf("False\n");
  #endif



  /* 1) Initialize CUBLAS */
  status = hipblasCreate(&handle);
  if (status != HIPBLAS_STATUS_SUCCESS){
    fprintf(stderr, "!!!! CUBLAS initialization error\n");
    return EXIT_FAILURE;
  }


  /* 2) Set math mode */
  printf("Compute Type...................%s\n\n", cublasComputeTypesStr[comptype]);
  //status = hipblasSetMathMode(handle, cublasMathModes[0]);
  if (status != HIPBLAS_STATUS_SUCCESS){
    fprintf(stderr, "!!!! CUBLAS MATH MODE ERROR\n");
    return EXIT_FAILURE;
  }


  /* 3) Allocate and fill host memory for the matrices */
  printf("Host mallocs A B C............."); fflush(stdout);
  t1 = omp_get_wtime();
  #ifdef PINNED
      hipHostMalloc((void**)&h_A, nelem*sizeof(h_A[0]));
      hipHostMalloc((void**)&h_B, nelem*sizeof(h_B[0]));
      hipHostMalloc((void**)&h_C, nelem*sizeof(h_C[0]));
  #else
      h_A = (ATYPE*)(malloc(nelem * sizeof(h_A[0])));
      h_B = (BTYPE*)(malloc(nelem * sizeof(h_B[0])));
      h_C = (CTYPE*)(malloc(nelem * sizeof(h_C[0])));
  #endif

  t2 = omp_get_wtime();
  printf("done: %f secs\n", t2-t1); fflush(stdout);
  printf("Filling matrices in Host......."); fflush(stdout);
  t1 = omp_get_wtime();
  fillMatrixRand<ATYPE>(h_A, nelem);
  fillMatrixRand<BTYPE>(h_B, nelem);
  fillMatrixRand<CTYPE>(h_C, nelem);
  t2 = omp_get_wtime();
  printf("done: %f secs\n", t2-t1); fflush(stdout);
  print_matrix<ATYPE>(h_A, N, N, "MAT A");
  print_matrix<BTYPE>(h_B, N, N, "MAT B");


  /* 4) Allocate device memory for the matrices */
  printf("Device mallocs A B C..........."); fflush(stdout);
  t1 = omp_get_wtime();
  if (hipMalloc(reinterpret_cast<void **>(&d_A), nelem * sizeof(d_A[0])) != hipSuccess) {
        fprintf(stderr, "!!!! device memory allocation error (allocate A)\n");
        return EXIT_FAILURE;
  }

  if (hipMalloc(reinterpret_cast<void **>(&d_B), nelem * sizeof(d_B[0])) != hipSuccess) {
    fprintf(stderr, "!!!! device memory allocation error (allocate B)\n");
    return EXIT_FAILURE;
  }

  if (hipMalloc(reinterpret_cast<void **>(&d_C), nelem * sizeof(d_C[0])) != hipSuccess) {
    fprintf(stderr, "!!!! device memory allocation error (allocate C)\n");
    return EXIT_FAILURE;
  }
  t2 = omp_get_wtime();
  printf("done: %f secs\n", t2-t1); fflush(stdout);



  /* 5) Initialize the device matrices with the host matrices */
  printf("Host -> Device memcpy A........"); fflush(stdout);
  t1 = omp_get_wtime();
  status = hipblasSetVector(nelem, sizeof(h_A[0]), h_A, 1, d_A, 1);
  if (status != HIPBLAS_STATUS_SUCCESS) {
    fprintf(stderr, "!!!! device access error (write A)\n");
    return EXIT_FAILURE;
  }
  gpuErrchk(hipDeviceSynchronize());
  t2 = omp_get_wtime();
  printf("done: %f secs (%f GB/sec)\n", t2-t1, nelem*sizeof(h_A[0])/(1e9 * (t2-t1))); fflush(stdout);

  printf("Host -> Device memcpy B........"); fflush(stdout);
  t1 = omp_get_wtime();
  status = hipblasSetVector(nelem, sizeof(h_B[0]), h_B, 1, d_B, 1);
  if (status != HIPBLAS_STATUS_SUCCESS) {
    fprintf(stderr, "!!!! device access error (write B)\n");
    return EXIT_FAILURE;
  }
  gpuErrchk(hipDeviceSynchronize());
  t2 = omp_get_wtime();
  printf("done: %f secs (%f GB/sec)\n", t2-t1, nelem*sizeof(h_B[0])/(1e9 * (t2-t1))); fflush(stdout);

  printf("Host -> Device memcpy C........"); fflush(stdout);
  t1 = omp_get_wtime();
  status = hipblasSetVector(nelem, sizeof(h_C[0]), h_C, 1, d_C, 1);
  if (status != HIPBLAS_STATUS_SUCCESS) {
    fprintf(stderr, "!!!! device access error (write C)\n");
    return EXIT_FAILURE;
  }
  gpuErrchk(hipDeviceSynchronize());
  t2 = omp_get_wtime();
  printf("done: %f secs (%f GB/sec)\n\n", t2-t1, nelem*sizeof(h_C[0])/(1e9 * (t2-t1))); fflush(stdout);







  /* 6) GEMM -> GPU CUBLAS */
  if(mode==1){
      printf("[CUBLAS] GPU GEMM.............."); fflush(stdout);
      gpuErrchk(hipEventRecord(start));
      status = hipblasGemmEx(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, N, N, &alpha,
                                        d_A, dtypeA, N,
                                        d_B, dtypeB, N,
                              &beta,    d_C, dtypeC, N, cublasComputeTypes[comptype],  CUBLAS_GEMM_DEFAULT_TENSOR_OP);
      if(status != HIPBLAS_STATUS_SUCCESS){
        fprintf(stderr, "!!!! kernel execution error.\n");
        return EXIT_FAILURE;
      }
      gpuErrchk(hipDeviceSynchronize());
      gpuErrchk(hipEventRecord(stop));
      gpuErrchk(hipEventSynchronize(stop));
      gpuErrchk(hipEventElapsedTime(&gputime_ms, start, stop));
      double gpuTFLOPS = TFLOP/(gputime_ms/1000.0);
      printf("done: %f secs [%f TFLOPS]\n", gputime_ms/1000.0, gpuTFLOPS); fflush(stdout);
  }





  /* 7) GEMM -> CPU BASIC */
  if(mode == 0){
      cblasC = cblas_compute<CPUTYPE>(N, nelem, alpha, beta, h_A, h_B, dtypeCPU, true); 
      /*
      //printf("[CBLAS] Host mallocs A B C............."); fflush(stdout);
      t1 = omp_get_wtime();
      cblasA = (CPUTYPE*)(malloc(nelem * sizeof(CPUTYPE)));
      cblasB = (CPUTYPE*)(malloc(nelem * sizeof(CPUTYPE)));
      cblasC = (CPUTYPE*)(malloc(nelem * sizeof(CPUTYPE)));
      t2 = omp_get_wtime();
      //printf("done: %f secs\n", t2-t1); fflush(stdout);
      //printf("[CBLAS] Filling matrices in Host......."); fflush(stdout);
      t1 = omp_get_wtime();
      copyMatrix<CPUTYPE, ATYPE>(cblasA, h_A, N);
      copyMatrix<CPUTYPE, BTYPE>(cblasB, h_B, N);
      t2 = omp_get_wtime();
      //printf("done: %f secs\n", t2-t1); fflush(stdout);
      t1 = omp_get_wtime();
      printf("[CBLAS] CPU GEMM (%6s)......", dtypeCPU); fflush(stdout);
      #ifdef CPUFP64
          cblas_dgemm(CblasColMajor,CblasNoTrans,CblasNoTrans,N,N,N,alpha,cblasA,N,cblasB,N,beta,cblasC,N);
      #else
          cblas_sgemm(CblasColMajor,CblasNoTrans,CblasNoTrans,N,N,N,alpha,cblasA,N,cblasB,N,beta,cblasC,N);
      #endif

      t2 = omp_get_wtime();
      double cpuTFLOPS = TFLOP/(t2-t1);
      printf("done: %f secs [%f TFLOPS]\n\n", t2-t1, cpuTFLOPS); fflush(stdout);
      print_matrix<CPUTYPE>(cblasC, N, N, "RESULT MAT C (CPU)");
      */
  }





  /* 8) Read the result back */
  if(mode == 1){
      printf("Device -> Host memcpy C........"); fflush(stdout);
      t1 = omp_get_wtime();
      status = hipblasGetVector(nelem, sizeof(h_C[0]), d_C, 1, h_C, 1);
      if (status != HIPBLAS_STATUS_SUCCESS) {
        fprintf(stderr, "!!!! device access error (read C)\n");
        return EXIT_FAILURE;
      }
      t2 = omp_get_wtime();
      printf("done: %f secs (%f GB/sec)\n", t2-t1, nelem*sizeof(h_C[0])/(1e9*(t2-t1))); fflush(stdout);
      print_matrix<CTYPE>(h_C, N, N, "RESULT MAT C (GPU)");
  }




  /* 9) Check result against reference */
  if(mode == 1){
      printf("Verify result.................."); fflush(stdout);
      t1 = omp_get_wtime();
      if(N < LIM_CHECK_N){
          cblasC = cblas_compute<CPUTYPE>(N, nelem, alpha, beta, h_A, h_B, dtypeCPU, false); 
      }
      double maxError = computeMaxError<CPUTYPE>(cblasC, h_C, N); 
      t2 = omp_get_wtime();
      printf("done: %f secs (maxError = %f%%, TOL = %f%%)\n%s\n\n", t2-t1,
              maxError*100.0, TOLERR*100.0, 
              maxError <= TOLERR ? (const char*)"pass" : (const char*) "failed"); fflush(stdout);
  }






  /* 10) Memory clean up */
  #ifdef PINNED
      hipHostFree(h_A);
      hipHostFree(h_B);
      hipHostFree(h_C);
  #else
      free(h_A);
      free(h_B);
      free(h_C);
  #endif

  if (hipFree(d_A) != hipSuccess) {
    fprintf(stderr, "!!!! memory free error (A)\n");
    return EXIT_FAILURE;
  }
  if (hipFree(d_B) != hipSuccess) {
    fprintf(stderr, "!!!! memory free error (B)\n");
    return EXIT_FAILURE;
  }
  if (hipFree(d_C) != hipSuccess) {
    fprintf(stderr, "!!!! memory free error (C)\n");
    return EXIT_FAILURE;
  }

  /* 11) Shutdown */
  status = hipblasDestroy(handle);
  if (status != HIPBLAS_STATUS_SUCCESS) {
    fprintf(stderr, "!!!! shutdown error (A)\n");
    return EXIT_FAILURE;
  }
}
